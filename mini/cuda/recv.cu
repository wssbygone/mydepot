#include "flowtranfer.hpp"
#include <thread>
#include <chrono>
#include <unistd.h>
#include <iostream>
#include <sys/time.h>

void buffertohandle(const char *buf,size_t buf_size,hipIpcMemHandle_t *my_handle)
{
    memcpy((char *)(my_handle), buf, sizeof(*my_handle));
}

void recv(const char *topic,const char *data,size_t size)
{
    char buf[128];
    char *data1;
    hipIpcMemHandle_t my_handle;

    buffertohandle(data,sizeof(hipIpcMemHandle_t),&my_handle);
    hipIpcOpenMemHandle((void **)&data1, my_handle, hipIpcMemLazyEnablePeerAccess);
    hipMemcpy(buf, data1, 64*sizeof(char), hipMemcpyDeviceToHost);
    //std::cout << data << std::endl;
    printf("%s \n",buf);
}
int main(int argc ,char **argv)
{
	FlowRecv *ptr = new FlowRecv("0.0.0.0","24014","camera0",&recv);
	while(1){
        sleep(1);
	}
}

