#include "flowtranfer.hpp"
#include <thread>
#include <chrono>
#include <unistd.h>
#include <iostream>
#include <sys/time.h>

#define DSIZE 128

void handletobuffer(hipIpcMemHandle_t my_handle, char *buf,size_t buf_size)
{
    memset(buf, 0, sizeof(my_handle)+1);
    memcpy(buf, (char *)(&my_handle), sizeof(my_handle));
}

int main(int argc ,char **argv)
{
	char buf[64];
    char handlebuf[128];

    char  *data;
    struct timeval tm;
    hipIpcMemHandle_t my_handle;
 	FlowSend *pptr = new FlowSend("0.0.0.0","24014");

	while(1){
        hipMalloc(&data, DSIZE*sizeof(char));
        hipMemset(data, 0, DSIZE*sizeof(char));
        hipIpcGetMemHandle(&my_handle, data);

        gettimeofday(&tm,NULL);
        sprintf(buf,"%ld: %ld \n",tm.tv_sec,tm.tv_usec);
        printf("%s \n",buf);
        hipMemcpy(data, buf, 64*sizeof(char), hipMemcpyHostToDevice);
        handletobuffer(my_handle,handlebuf,sizeof(handlebuf));

 		pptr->send("camera0",static_cast<char*>(handlebuf),sizeof(handlebuf));
        sleep(1);
	}
}

